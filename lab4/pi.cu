#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void vectorAdd(int *result,int N,int threadNum,int blockNum){
    double sum = 0.0,t = 1.0/N;
    int i = 0;
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    for(i=bid*threadNum+tid;i<N;i+=threadNum*blockNum){
        sum += 1/((1+((i+1)*t+i*t)/2*((i+1)*t+i*t)/2));
    }
    result[bid*threadNum + tid] = 4*t*sum;
}
int main(int argc,char **argv){
    //initial
    int N = 50,threadNum = 4,blockNum = 4;
    if(argc>1) N = atoi(argv[1]);
    if(argc>2) threadNum = atoi(argv[2]);
    if(argc>3) blockNum = atoi(argv[3]);
    clock_t start;
    //allocate device mem
    double *cuda_result;
    start = clock();
    cudeMalloc((void*)&cuda_result,sizeof(double)*blockNum*threadNum);
    //cal
    vectorAdd<<<blockNum,threadNum,0>>>(cuda_result,N,threadNum,blockNum);
    //device to host
    int sum[threadNum*blockNum];
    cudaMemcpu(&sum,cuda_result,sizeof(int)*threadNum*blockNum,hipMemcpyDeviceToHost);
    cudeFree(cuda_result);
    double final_sum = 0;
    int i ;
    for(i=0;i<threadNum*blockNum;i++){
        final_sum +=sum[i];
    }
    printf("divide:%d,threadNum:%d,blockNum:%d; pi = %.16f,time =
            %f\n",N,threadNum,blockNum,final_sum,clock()-start);
    return 0;
}
